// A100 PCIE 80GB
// Testing iters = 200.
// Test performance using shape M=5376, N=5376, K=2048
// Running cost of CUDA kernel is 4.46636ms
// TFLOPS: 26.5048

#include <hip/hip_fp16.h>
#include <mma.h>
#include <hip/hip_runtime.h>

const int MI = 128;
// const int NI = 128;
const int KI = 32;
const int MII = 64;
const int NII = 64;
const int KII = 16;
const int wmmaM = 16;
const int wmmaN = 16;
const int wmmaK = 16;
#define ZZ(x) 

__device__ void loadSmemA(half *smem, half *A, int M, int K, int ko) // 第几个 ko 代表 row 上第几个 tile
{ // A[M, K]
    // load 128 * 32
    // load MI * KI
    int by = blockIdx.y; // 这个绝定,  所以是 [128, 32] // 用 128个 thread 来 load
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int tz = threadIdx.z; // 32, 2, 2  // 128
    int tid = tz * 64 + ty * 32 + tx;
    for (int i = 0; i < 32; ++i) // 32 个数
    {
        int row = i * 4 + tid / 32;
        int col = tid % 32;
        // layout: [row_out, col_out, row_in, col_in] = [8, 2, 16, 16]
        // auto s_elem = smem[row / 16, col / 16, row % 16, col % 16]; // [8, 2, 16, 16];
        // auto a_elem = A[by * 128 + row, ko * KI + col]; // [M, K]
        smem[row / 16 * (2 * 16 * 16) + col / 16 * (16 * 16) + row % 16 * 16 + col % 16] = A[(by * 128 + row) * K + ko * KI + col];
    }
}

__device__ void loadSmemB(half *smem, half *B, int N, int K, int ko)
{ // B[N, K]
    // load 128 * 32
    int bx = blockIdx.x;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int tz = threadIdx.z;
    int tid = tz * 64 + ty * 32 + tx;
    for (int i = 0; i < 32; ++i)
    {
        int row = i * 4 + tid / 32;
        int col = tid % 32;
        // layout: [row_out, col_out, row_in, col_in] = [8, 2, 16, 16]
        // auto B_elem = B[bx * 128 + row, ko * KI + col];
        // auto s_elem = smem[row / 16,col / 16, row % 16, col % 16];

        smem[row / 16 * (2 * 16 * 16) + col / 16 * (16 * 16) + row % 16 * 16 + col % 16] = B[(bx * 128 + row) * K + ko * KI + col];
    }
}

__device__ void loadSmemC(float *smem, half *C, int M, int N)
{
    // load 128 * 128
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int tz = threadIdx.z;
    int tid = tz * 64 + ty * 32 + tx;
    for (int i = 0; i < 128; ++i)
    {
        int row = i;
        int col = tid;
        // layout: [row_out, col_out, row_in, col_in] = [8, 8, 16, 16]
        smem[row / 16 * (8 * 16 * 16) + col / 16 * (16 * 16) + row % 16 * 16 + col % 16] = (float)(C[(by * 128 + row) * N + bx * 128 + col]);
    }
}

__device__ void storeSmemC(half *C, float *smem, int M, int N)
{
    // load 128 * 128
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int tz = threadIdx.z;
    int tid = tz * 64 + ty * 32 + tx;
    for (int i = 0; i < 128; ++i)
    {
        int row = i;
        int col = tid;
        // layout: [row_out, col_out, row_in, col_in] = [8, 8, 16, 16]
        (C[(by * 128 + row) * N + bx * 128 + col]) = (half)smem[row / 16 * (8 * 16 * 16) + col / 16 * (16 * 16) + row % 16 * 16 + col % 16];
    }
}

__device__ void loadFragA(nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, wmmaM, wmmaN, wmmaK, half, nvcuda::wmma::row_major> *frag, half *smem, int ki)
{
    // load 64x16
    int tz = threadIdx.z;
    // smem: [8, 2, 16, 16]
    for (int i = 0; i < 4; ++i)
    {
        int row = tz * 64 + i * 16; // 0: [0:3, 0, 0, 0] 1: [4:7, 0, 0, 0]
        int col = ki * KII;  // [0:3, 1, 0, 0]; 这个是下一轮了
        // auto s = smem[row /16, col / 16, 0, 0];
        nvcuda::wmma::load_matrix_sync(frag[i], smem + row / 16 * (2 * 16 * 16) + col / 16 * (16 * 16), 16);
    }
}

__device__ void loadFragB(nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, wmmaM, wmmaN, wmmaK, half, nvcuda::wmma::col_major> *frag, half *smem, int ki)
{
    // load 64x16
    int ty = threadIdx.y;
    for (int i = 0; i < 4; ++i)
    {
        int row = ty * 64 + i * 16;
        int col = ki * KII;
        nvcuda::wmma::load_matrix_sync(frag[i], smem + row / 16 * (2 * 16 * 16) + col / 16 * (16 * 16), 16);
    }
}

__device__ void storeAccum(float *ptr, nvcuda::wmma::fragment<nvcuda::wmma::accumulator, wmmaM, wmmaN, wmmaK, float> *frag)
{
    // store 64x64
    int ty = threadIdx.y;
    int tz = threadIdx.z;
    for (int i = 0; i < 4; ++i)
    {
        for (int j = 0; j < 4; ++j)
        {
            int row = tz * 64 + i * 16;
            int col = ty * 64 + j * 16;
            // laoyut: [8, 8, 16, 16]
            nvcuda::wmma::store_matrix_sync(ptr + row / 16 * (8 * 16 * 16) + col / 16 * (16 * 16), frag[i * 4 + j], 16, nvcuda::wmma::mem_row_major);
        }
    }
}

__global__ void matmul(half *A, half *B, half *C, int M, int N, int K, float alpha, float beta)
{
    // A is row-major
    // B is col-major
    // 128 threads [x, y, z] = [32, 2, 2]
    // threadblock mma: 128x128x32
    // warp mma: 64x64x16
    extern __shared__ uint8_t shared_storage[];
    half *SA = reinterpret_cast<half *>(shared_storage); // TODO: SA大小?  [MI(128), KI(32)]
    half *SB = reinterpret_cast<half *>(shared_storage + MI * KI * sizeof(half)); // TODO: SB大小?
    float *SC = reinterpret_cast<float *>(shared_storage); // SC 好像是共用的? 

    nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, wmmaM, wmmaN, wmmaK, half, nvcuda::wmma::row_major> FragA[MII / wmmaM];
                                                /* 16,    16,    16                                          64  /    16  */
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, wmmaM, wmmaN, wmmaK, half, nvcuda::wmma::col_major> FragB[NII / wmmaN];
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, wmmaM, wmmaN, wmmaK, float> Accum[MII / wmmaM * NII / wmmaN];
                                                                                    /*  64  /    16 *  64 /    16 */

    for (int mii = 0; mii < MII / wmmaM  /* 4 */; mii += 1)
    {
        for (int nii = 0; nii < NII / wmmaN  /* 4 */; nii += 1)
        {
            nvcuda::wmma::fill_fragment(Accum[mii * (NII / wmmaN) + nii], 0.0);
        }
    }
    for (int ko = 0; ko < K / KI; ko += 1)
    {
        loadSmemA(SA, A, M, K, ko); // [128, 32]
        loadSmemB(SB, B, N, K, ko); // [128, 32]
        __syncthreads();
        for (int ki = 0; ki < KI / KII /* 2 = 32 / 16*/; ki += 1)
        {
            // 64x64x16 mma for each warp
            loadFragA(FragA /*16 * 16 * 4*/, SA, ki); // 分了两个tile, 每个人 [8, 1, 16, 16]
            loadFragB(FragB, SB, ki);
            for (int mii = 0; mii < MII / wmmaM; mii += 1)
            {
                for (int nii = 0; nii < NII / wmmaN; nii += 1)
                {
                    // 16x16x16 for each wmma
                    nvcuda::wmma::mma_sync(Accum[mii * (NII / wmmaN) + nii], FragA[mii], FragB[nii], Accum[mii * (NII / wmmaN) + nii]);
                }
            }
        }
    }
    storeAccum(SC, Accum);
    __syncthreads();
    storeSmemC(C, SC, M, N);
}
